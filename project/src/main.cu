#include "hip/hip_runtime.h"
#include "debug.hpp"
#include "cuda.hpp"

namespace IMAGE {

	const u8 CHANNELS = 3; // Hack! We're hardcodding image channels to 3!

	void Load (
		/* IN  */ const u8* image,
		/* IN  */ const u32& imageCount,
		/* OUT */ r32*& output
	) {
		output = new r32[imageCount / CHANNELS];

		s32 sum = 0;

		for (u32 i = 0; i < imageCount; ++i) {
			sum += image[i];
			if ((i + 1) % CHANNELS == 0) {
				output[i / CHANNELS] = (sum / 3.f) / 255.f;
				sum = 0;
			}
		}
	}

	void Save (
		/* IN  */ const r32* image,
		/* IN  */ const u32& imageCount,
		/* OUT */ u8*& output
	) {
		output = new u8[imageCount * CHANNELS];

		for (u32 i = 0; i < imageCount; ++i) {
			for (u32 j = 0; j < CHANNELS; ++j) {
				output[i * CHANNELS + j] = image[i] * 255;
			}
		}
	}

}


namespace GPU {

	const u8 BLOCK_SIZE = 32;

	__global__ void RobertsFilter (
		/* IN  */ const r32*    iPixels,    //
		/* OUT */ r32*          oPixels,    //
		/* IN  */ s32           width,      //
		/* IN  */ s32           height,     //
		/* IN  */ s32           strength,   //
		/* IN  */ s32           mode        //
	) {
		u32 x = blockIdx.x * blockDim.x + threadIdx.x;
		u32 y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width - 1 && y < height - 1) {
			r32 gx = iPixels[y * width + x] - iPixels[(y + strength) * width + (x + strength)];
			r32 gy = iPixels[y * width + (x + strength)] - iPixels[(y + strength) * width + x];

			if (mode == 2) gy = 0.0;                            // GX only
			if (mode == 3) gx = 0.0;                            // GY only

			r64 magnitude = sqrtf ((r32)(gx * gx + gy * gy));

			if (mode == 4) magnitude = 1 - magnitude;           // Inverse mode

			oPixels[y * width + x] = (r32)(magnitude);
		}

	}

}


s32 main() {

	DEBUG (DEBUG_TYPE_UNKNOWN) LogInfo ("Entered Roberts Cross execution!");
	
	s32 width;
	s32 height;
	s32 rgb;
	u8* image;
	u32 imageSize;

	r32* cpuI;
	r32* cpuO;
	r32* gpuI;
	r32* gpuO;

	s32 mode;
	s32 fileId;
	
	s32 strength (1); // default
	

	{ // Menu creation, input read, validation

		printf (
			"Select Mode:\n"
			" -> 1. both\n"
			" -> 2. gx\n"
			" -> 3. gy\n"
			" -> 4. inverse\n"
			" : "
		);
		scanf ("%i", &mode);

		printf ("Select file (0-4): ");
		scanf ("%i", &fileId);
	
		printf ("Algorithm strength: ");
		scanf ("%i", &strength);

		putc ('\n', stdout);

		if (mode > 4 || mode < 1) { mode = 1; }
		if (fileId > 4 || fileId < 0) { fileId = 0; }

	}

	{ // Loading image
		DEBUG (DEBUG_TYPE_UNKNOWN) LogInfo ("Loading Image");

		auto&& filePath = files[fileId];
		image = stbi_load (filePath, &width, &height, &rgb, IMAGE::CHANNELS);

		DEBUG (DEBUG_TYPE_UNKNOWN) if (image == nullptr) {
			Error ("Could not load the image.");
		}

		IMAGE::Load (image, width * height * IMAGE::CHANNELS, cpuI);
		imageSize = width * height * sizeof (u32);
	}

	{ // CPU allocations
		cpuO = (r32*) malloc (imageSize);
	}
	
	{ // GPU allocations
		hipMalloc ((void**)&gpuI, imageSize);
		hipMalloc ((void**)&gpuO, imageSize);
		hipMemcpy (gpuI, cpuI, imageSize, hipMemcpyHostToDevice);
	}
	
	{ // Processing
		dim3 blockSize (GPU::BLOCK_SIZE, GPU::BLOCK_SIZE, 1);

		dim3 gridSize (
			(width + blockSize.x - 1) / blockSize.x,
			(height + blockSize.y - 1) / blockSize.y,
			1
		);

		DEBUG (DEBUG_TYPE_UNKNOWN) LogInfo ("Processing");

		GPU::RobertsFilter <<<gridSize, blockSize>>> (gpuI, gpuO, width, height, strength, mode);

		hipMemcpy (cpuO, gpuO, imageSize, hipMemcpyDeviceToHost);
	}
	
	{ // Outputting
		DEBUG (DEBUG_TYPE_UNKNOWN) LogInfo ("Outputting");
	
		// We're reusing this memory...
		stbi_image_free (image);
	
		IMAGE::Save (cpuO, width * height, image);
	
		stbi_write_png (
			OF, width, height, IMAGE::CHANNELS,
			image, width * IMAGE::CHANNELS
		);
	}
	
	{ // Freeing
		hipFree (gpuI);
		hipFree (gpuO);
		free (cpuI);
		free (cpuO);
	}

	DEBUG (DEBUG_TYPE_UNKNOWN) LogInfo ("Finished execution.");

	return 0;
}
